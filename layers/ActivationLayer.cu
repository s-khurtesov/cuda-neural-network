#include "ActivationLayer.cuh"

ActivationLayer::ActivationLayer(std::string name_, LayerShape shape_, hipdnnHandle_t hCudnn_,
	hipdnnActivationMode_t activationMode_,
	hipdnnNanPropagation_t reluNanOpt_,
	double coef_) : hCudnn(hCudnn_), activationMode(activationMode_), reluNanOpt(reluNanOpt_), 
	coef(coef_), activationDesc(NULL)
{
	this->name = name_;
	this->shape = shape_;

	x.init(shape.batch_size, shape.in_nrns, shape.in_nrn_h, shape.in_nrn_w);
	dx = x;

	x.fill(0.0f);
	dx.fill(0.0f);

	initActivationDesc();
}

void ActivationLayer::initActivationDesc()
{
	CHECK_CUDNN(hipdnnCreateActivationDescriptor(&activationDesc));
	CHECK_CUDNN(hipdnnSetActivationDescriptor(activationDesc, activationMode, reluNanOpt, coef));
}

void ActivationLayer::init() { }

void ActivationLayer::forward()
{
	CHECK_CUDNN(hipdnnActivationForward(hCudnn, activationDesc, alpha, x.desc, x.data, beta, y->desc, y->data));
}

void ActivationLayer::backward(float learning_rate, bool last)
{
	CHECK_CUDNN(hipdnnActivationBackward(hCudnn, activationDesc, alpha, y->desc, y->data, dy->desc, dy->data, x.desc, x.data, beta, dx.desc, dx.data));
}

ActivationLayer::~ActivationLayer()
{
	if (activationDesc) {
		CHECK_CUDNN(hipdnnDestroyActivationDescriptor(activationDesc));
	}
}
