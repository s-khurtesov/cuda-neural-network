#include "hip/hip_runtime.h"
#include "NeuralNetwork.cuh"

extern __device__ float atomicAdd(float* address, float val);

void NeuralNetwork::init()
{
	assert(!initialized);
	assert(layers.size() >= 1);

	Tensor* tmp_x = layers.front()->getX();
	LayerShape last_shape = layers.back()->getShape();

	x.init(tmp_x->N, tmp_x->C, tmp_x->H, tmp_x->W, tmp_x->format);
	y.init(last_shape.batch_size, last_shape.out_nrns, last_shape.out_nrn_h, last_shape.out_nrn_w);
	dy = y;

	x.fill(0);
	y.fill(0);
	dy.fill(0);

	for (auto iter = layers.begin(); iter != layers.end() - 1; iter++) {
		(*iter)->setY((*(iter + 1))->getX());
		(*iter)->setdY((*(iter + 1))->getdX());
	}
	layers.back()->setY(&y);
	layers.back()->setdY(&dy);

	for (Layer* cur_layer : layers) {
		cur_layer->init();
	}

	initialized = true;
}

void NeuralNetwork::clear()
{
	assert(initialized);

	layers.clear();

	initialized = false;
}

void NeuralNetwork::addLayer(Layer* p_layer)
{
	assert(!initialized);

	layers.push_back(p_layer);
}

Tensor& NeuralNetwork::forward(Tensor& x)
{
	assert(initialized);

	layers.front()->setX(x);
	for (Layer* cur_layer : layers) {
		cur_layer->forward();
	}
	CHECK_CUDA(hipDeviceSynchronize());
	return y;
}

void NeuralNetwork::backward(Tensor& dy, float learning_rate)
{
	assert(initialized);

	this->dy = dy;
	for (auto iter = layers.rbegin(); iter != layers.rend(); iter++) {
		(*iter)->backward(learning_rate, *iter == layers.front());
	}
	CHECK_CUDA(hipDeviceSynchronize());
}

__global__ void binaryCrossEntropyCost(float* predictions, float* target, int size, float* cost) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size) {
		float partial_cost = target[index] * logf(predictions[index])
			+ (1.0f - target[index]) * logf(1.0f - predictions[index]);
		atomicAdd(cost, -partial_cost / size);
	}
}

__global__ void dBinaryCrossEntropyError(float* predictions, float* target, int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size) {
		predictions[index] = -1.0f * (target[index] / predictions[index]
			- (1.0f - target[index]) / (1.0f - predictions[index]));
	}
}

void NeuralNetwork::calcError(Tensor& labels)
{
	dim3 block_size(128);
	dim3 num_of_blocks((dy.size() + block_size.x - 1) / block_size.x);
	dy = y;
	dBinaryCrossEntropyError<<<num_of_blocks, block_size>>>(
		dy.data,
		labels.data,
		dy.size());
	CHECK_CUDA(hipGetLastError());
}

void NeuralNetwork::calcCost(Tensor& labels, float* cost)
{
	dim3 block_size(128);
	dim3 num_of_blocks((y.size() + block_size.x - 1) / block_size.x);
	binaryCrossEntropyCost<<<num_of_blocks, block_size>>>(
		y.data,
		labels.data,
		y.size(), cost);
	CHECK_CUDA(hipDeviceSynchronize());
	CHECK_CUDA(hipGetLastError());
}

void NeuralNetwork::train(Tensor& x, Tensor& labels, int iters, float learning_rate, float learning_rate_lowering_coef)
{
	assert(initialized);
	assert(y.N == labels.N);
	assert(y.C == labels.C);
	assert(y.H == labels.H);
	assert(y.W == labels.W);
	assert(y.format == labels.format);

	float cur_learning_rate = learning_rate;
	float* cost;
	int period = iters / 10;
	int short_period = iters / 100;
	float lr_decrement = (1.0f - learning_rate_lowering_coef) * learning_rate / iters;

	if (!period)
		period = 1;
	if (!short_period)
		short_period = 1;

	CHECK_CUDA(hipMallocManaged(&cost, sizeof(float)));

	layers.front()->setX(x);

	CHECK_CUDA(hipDeviceSynchronize());

	for (int iteration = 0; iteration < iters; iteration++) {
		for (Layer* cur_layer : layers) {
			cur_layer->forward();
		}

		calcError(labels);
		
		for (auto iter = layers.rbegin(); iter != layers.rend(); iter++) {
			(*iter)->backward(learning_rate, *iter == layers.front());
		}

		calcCost(labels, cost);

		if ((iteration < period && (iteration + 1) % (short_period) == 0) || (iteration + 1) % (period) == 0) {
			printf("Iteration: %d, Cost: %f, learning_rate: %f, y_int: %d, dy_nan: %d, dy_inf: %d\n", iteration + 1, *cost, cur_learning_rate, 
				std::count_if(y.data, y.data + y.size() - 1, [](float x) {return (int)x == (float)x; }), 
				std::count_if(dy.data, dy.data + dy.size() - 1, [](float x) {return isnan(x); }), 
				std::count_if(dy.data, dy.data + dy.size() - 1, [](float x) {return isinf(x); }));
		}
		*cost = 0.0f;
		cur_learning_rate = cur_learning_rate - lr_decrement;
	}

	CHECK_CUDA(hipDeviceSynchronize());

	CHECK_CUDA(hipFree(cost));
}
