#include "common.cuh"

hipdnnHandle_t g_hCudnn;

int main()
{
    InitCuda();
    InitCudnn(&g_hCudnn);

    CleanCudnn(&g_hCudnn);
    CleanCuda();

    return 0;
}
