#include "common.cuh"

#include <stdlib.h>
#include <windows.h>
#include <string>

std::string GetErrorString(DWORD errorMessageID)
{
    LPSTR messageBuffer = nullptr;
    size_t size = FormatMessageA(FORMAT_MESSAGE_ALLOCATE_BUFFER | FORMAT_MESSAGE_FROM_SYSTEM | FORMAT_MESSAGE_IGNORE_INSERTS,
        NULL, errorMessageID, MAKELANGID(LANG_ENGLISH, SUBLANG_DEFAULT), (LPSTR)&messageBuffer, 0, NULL);

    std::string message(messageBuffer, size);

    //Free the buffer.
    LocalFree(messageBuffer);

    return message;
}

const char* __stdcall cublasGetErrorString(hipblasStatus_t hipblasStatus_t)
{
    switch (hipblasStatus_t) {
    case HIPBLAS_STATUS_SUCCESS:
        return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:
        return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
        return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
        return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
        return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
        return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
        return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
        return "HIPBLAS_STATUS_INTERNAL_ERROR";
    case HIPBLAS_STATUS_NOT_SUPPORTED:
        return "HIPBLAS_STATUS_NOT_SUPPORTED";
    case HIPBLAS_STATUS_UNKNOWN:
        return "HIPBLAS_STATUS_UNKNOWN";
    default:
        return "Undefined cuBLAS status";
    }
}

void Check(errno_t err, errno_t success, const char* descr, const char* file, const int line)
{
    DWORD dErr = GetLastError();
    if (err != success) {
        std::string msg = GetErrorString(dErr);
        fprintf(stderr, "%s:%-4d %s\nERROR #%d (returned %d): %s", strrchr(file, '/') + 1, line, descr, dErr, err, msg.c_str());
        CleanCuda();
        throw std::runtime_error(msg.c_str());
    }
    else if (dErr != 0) {
        fprintf(stderr, "%s:%-4d WARNING: Returned as expected, but error #%d (returned %d): %s\n", strrchr(__FILE__, '/') + 1, __LINE__, dErr, err, GetErrorString(dErr).c_str());
    }
}

void CheckCuda(hipError_t cudaStatus, const char* descr, const char* file, const int line)
{
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "%s:%-4d %s\nERROR #%d: %s", strrchr(file, '/') + 1, line, descr, cudaStatus, hipGetErrorString(cudaStatus));
        CleanCuda();
        throw std::runtime_error(hipGetErrorString(cudaStatus));
    }
}

void CheckCublas(hipblasStatus_t hipblasStatus_t, const char* descr, const char* file, const int line)
{
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "%s:%-4d %s\nERROR #%d: %s", strrchr(file, '/') + 1, line, descr, hipblasStatus_t, cublasGetErrorString(hipblasStatus_t));
        CleanCuda();
        throw std::runtime_error(cublasGetErrorString(hipblasStatus_t));
    }
}

void CheckCudnn(hipdnnStatus_t cudnnStatus, const char* descr, const char* file, const int line)
{
    if (cudnnStatus != HIPDNN_STATUS_SUCCESS) {
        fprintf(stderr, "%s:%-4d %s\nERROR #%d: %s", strrchr(file, '/') + 1, line, descr, cudnnStatus, hipdnnGetErrorString(cudnnStatus));
        CleanCuda();
        throw std::runtime_error(hipdnnGetErrorString(cudnnStatus));
    }
}

errno_t JustCheck(errno_t err, errno_t success, const char* descr, const char* file, const int line)
{
    DWORD dErr = GetLastError();
    if (err != success) {
        fprintf(stderr, "%s:%-4d %s\nERROR #%d (returned %d): %s", strrchr(file, '/') + 1, line, descr, dErr, err, GetErrorString(dErr).c_str());
        return 1;
    }
    else if (dErr != 0) {
        fprintf(stderr, "%s:%-4d WARNING: Returned as expected, but error #%d (returned %d): %s\n", strrchr(__FILE__, '/') + 1, __LINE__, dErr, err, GetErrorString(dErr).c_str());
        return 2;
    }
    return 0;
}

errno_t JustCheckCuda(hipError_t cudaStatus, const char* descr, const char* file, const int line)
{
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "%s:%-4d %s\nERROR #%d: %s", strrchr(file, '/') + 1, line, descr, cudaStatus, hipGetErrorString(cudaStatus));
        return 1;
    }
    return 0;
}

errno_t JustCheckCublas(hipblasStatus_t hipblasStatus_t, const char* descr, const char* file, const int line)
{
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "%s:%-4d %s\nERROR #%d: %s", strrchr(file, '/') + 1, line, descr, hipblasStatus_t, cublasGetErrorString(hipblasStatus_t));
        return 1;
    }
    return 0;
}

errno_t JustCheckCudnn(hipdnnStatus_t cudnnStatus, const char* descr, const char* file, const int line)
{
    if (cudnnStatus != HIPDNN_STATUS_SUCCESS) {
        fprintf(stderr, "%s:%-4d %s\nERROR #%d: %s", strrchr(file, '/') + 1, line, descr, cudnnStatus, hipdnnGetErrorString(cudnnStatus));
        return 1;
    }
    return 0;
}

errno_t InitCuda()
{
    // Choose which GPU to run on, change this on a multi-GPU system.
    CHECK_CUDA(hipSetDevice(0));

    return 0;
}

errno_t CleanCuda()
{
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.

    CHECK_CUDA_EX(hipDeviceReset(), return 1);

    return 0;
}

errno_t InitCublas(hipblasHandle_t* handleCublas)
{
    hipblasCreate(handleCublas);

    return 0;
}

errno_t CleanCublas(hipblasHandle_t* handleCublas)
{

    return 0;
}

errno_t InitCudnn(hipdnnHandle_t* handleCudnn)
{
    CHECK_CUDNN(hipdnnCreate(handleCudnn));

    return 0;
}

errno_t CleanCudnn(hipdnnHandle_t* handleCudnn)
{
    CHECK_CUDNN(hipdnnDestroy(*handleCudnn));

    return 0;
}
